

#include <hip/hip_runtime.h>
#include <stdio.h>
/* #include <stdlib.h> */ /* malloc() */
#include <time.h>


//https://developer.nvidia.com/blog/even-easier-introduction-cuda

// make it a CUDA kernel function using __global__ keyword
//
__global__ 
void add_on_gpu ( int lim 
		, float *xaxis 
		, float *yaxis
		, float *sumxy
		
		){

	int idx ;

	//int trdidx = blockIdx.x * blockDim.x + threadIdx.x ;
	//int stride = blockDim.x * gridDim.x ;

	//int trdidx = threadIdx.x ;
	//int stride = blockDim.x ;

	//for ( idx = trdidx ; idx < lim ; idx += stride ){

	for ( idx = 0; idx < lim ; idx += 1 ){

		sumxy[idx] = xaxis[idx] + yaxis[idx] ;

	}

}

void add_on_cpu ( int lim 
		, float *xaxis 
		, float *yaxis
		, float *sumxy
		
		){

	int idx;

	for ( idx = 0 ; idx < lim ; idx ++ ){

		sumxy[idx] = xaxis[idx] + yaxis[idx] ;

	}

}

void printFloatArray( int lim
		, float *arr
		
		){
	int idx;

	for ( idx = 0 ; idx < lim ; idx ++ ) {
		printf("arr[idx] = %f\n", arr [ idx ] );
	}
}

void initializeData ( int lim
		, float *xaxis
		, float *yaxis
		, float *sumxy

		){

	int idx ;

	for ( idx = 0 ; idx < lim ; idx ++ ) {

		xaxis[idx] = 1.0;
		yaxis[idx] = 1.0;
		sumxy[idx] = 0.0;
	}

}

int main(void){

	int idx 
	    , lim  = 1e8 // data size - max 9 for int
	    // 9 will compile but wont run, max 8
	    , loopCnt = 1e7
	    , threadCnt = 1 // geforce gt 1030 has 384 cores - no difference after 2000 
	    , blockCnt = 1 //( lim + threadCnt - 1 ) / threadCnt // pretty much does not work so leave it at one
	    ;

	printf("lim: %d , loopCnt: %d \n", lim , loopCnt);

	float *xaxis ; 
	float *yaxis ; 
	float *sumxy ; 

	//xaxis = malloc( sizeof(float) * lim );
	//yaxis = malloc( sizeof(float) * lim );
	//sumxy = malloc( sizeof(float) * lim );

	// Allocate Unified Memory -- accessible from CPU or GPU
	hipMallocManaged( &xaxis , lim * sizeof(float) ) ;
	hipMallocManaged( &yaxis , lim * sizeof(float) ) ;
	hipMallocManaged( &sumxy , lim * sizeof(float) ) ;

	// time since jan 1st 1970
	time_t ct, lt, dt;
	ct = time(NULL); lt = ct; dt = 0;	

	printf("starting timer\n");
	printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );
	lt = ct; ct = time(NULL); dt = ct - lt; printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );

	
	// initialize data
	printf("\ninitializing data for add on cpu\n");
	initializeData( lim , xaxis , yaxis , sumxy ) ;
	lt = ct; ct = time(NULL); dt = ct - lt; printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );

	// add on cpu
	/*
	printf("adding on cpu\n");
	for ( idx = 0 ; idx < loopCnt ; idx ++ ) {
		add_on_cpu ( lim , xaxis , yaxis , sumxy ) ;
	}
	lt = ct; ct = time(NULL); dt = ct - lt; printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );
	*/

	// re-initialize data
	printf("\nre-initializing data for add on gpu\n");
	initializeData( lim , xaxis , yaxis , sumxy ) ;
	lt = ct; ct = time(NULL); dt = ct - lt; printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );

	//add on gpu
	printf("adding on gpu\n");
	for ( idx = 0 ; idx < loopCnt ; idx ++ ) {
		add_on_gpu <<< blockCnt , threadCnt >>> ( lim , xaxis , yaxis , sumxy ) ;
	}
	// wait for device(gpu) to finish before accessing on host(cpu)
	hipDeviceSynchronize() ;
	lt = ct; ct = time(NULL); dt = ct - lt; printf("lt: %d , ct: %d , dt: %d\n", lt, ct, dt );
	
	//printf("printing out data\n"); printFloatArray( lim , sumxy ) ;


	hipFree( xaxis ) ;
	hipFree( yaxis ) ;
	hipFree( sumxy ) ;


	return 0 ;
}
